#include "hip/hip_runtime.h"
#include "imp.cuh"
#include <iostream>
#include <ctime>
#include <random>

using namespace std;

void tests(int N){
  float *A = new float[N];
  float *B = new float[N];
  float *answerGPU = new float();
  float *answerGGPU = new float();
  *answerGPU = 0;
  *answerGGPU = 0;

  float mean = 0.0;      
  float stddev = 2.0;

  random_device rd;
  mt19937 gen(rd()); 
  normal_distribution<double> dist(mean, stddev);

  for (int i = 0; i < N; ++i) {
      A[i] = dist(gen);
      B[i] = dist(gen);
  }


  float answerCPU = CPUimplementation(A, B, N); // Вычисления на CPU

  float *cudaA, *cudaB; // Вычисления на GPU
  hipMalloc(&cudaA, N * sizeof(float));
  hipMalloc(&cudaB, N * sizeof(float));
  hipMalloc(&answerGPU, sizeof(float));

  hipMemcpy(cudaA, A, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(cudaB, B, N * sizeof(float), hipMemcpyHostToDevice);

  const int block_size = 256;
  int number_of_blocks = N/block_size + 1;

  GPUimplementation<<<number_of_blocks, block_size>>>(cudaA, cudaB, answerGPU);
  hipDeviceSynchronize();

  hipMemcpy(answerGGPU, answerGPU, sizeof(float), hipMemcpyDeviceToHost);

  // Сравнение результатов
  cout << "CPU: " << answerCPU << " GPU: " << *answerGGPU << endl;
  if (abs(answerCPU - *answerGGPU) < 1e-4) {
      cout << "Результаты совпадают!" << endl;
  } else {
      cout << "Результаты не совпадают!" << endl;
  }

  // Освобождение ресурсов
  delete[] A;
  delete[] B;
  delete answerGPU;
  delete answerGGPU;
  hipFree(cudaA);
  hipFree(cudaB);
  hipFree(answerGPU);
}

int main(){
  srand(time(0));
  for(int i = 1; i < 100; i++){
    cout << "Test " << i << ":" << endl;
    tests(1000000 + 25000 * i);
  }
}
