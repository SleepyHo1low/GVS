#include "hip/hip_runtime.h"
#include "imp.cuh"
#include <iostream>
#include <ctime>
#include <random>
#include <vector>

using namespace std;

void tests(int N, vector<string> &results){
  float *A = new float[N];
  float *B = new float[N];
  float *answerGPU = new float();
  float *answerGGPU = new float();
  *answerGPU = 0;
  *answerGGPU = 0;

  float mean = 0.0;      
  float stddev = 1.0;

  random_device rd;
  mt19937 gen(rd()); 
  normal_distribution<double> dist(mean, stddev);

  for (int i = 0; i < N; ++i) {
      A[i] = dist(gen);
      B[i] = dist(gen);
  }

  /*for (int i = 0; i < N; ++i) {
        A[i] = static_cast<float>(rand()) / RAND_MAX;
        B[i] = static_cast<float>(rand()) / RAND_MAX;
  }*/
  float answerCPU = CPUimplementation(A, B, N); // Вычисления на CPU

  float *cudaA, *cudaB; // Вычисления на GPU
  hipMalloc(&cudaA, N * sizeof(float));
  hipMalloc(&cudaB, N * sizeof(float));
  hipMalloc(&answerGPU, sizeof(float));

  hipMemcpy(cudaA, A, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(cudaB, B, N * sizeof(float), hipMemcpyHostToDevice);

  const int block_size = 256;
  int number_of_blocks = (N + block_size - 1) / block_size;

  GPUimplementation<<<number_of_blocks, block_size>>>(cudaA, cudaB, answerGPU);
  hipDeviceSynchronize();

  hipMemcpy(answerGGPU, answerGPU, sizeof(float), hipMemcpyDeviceToHost);
  
  string result = "CPU: " + to_string(answerCPU) + " GPU: " + to_string(*answerGGPU);
  if (abs(answerCPU - *answerGGPU) < 1e-2) {
      result += " | Результаты совпадают!";
  } else {
      result += " | Результаты не совпадают!";
  }
  results.push_back(result);

  // Освобождение ресурсов
  delete[] A;
  delete[] B;
/*
  delete answerGPU;
  delete answerGGPU;

  hipFree(cudaA);
  hipFree(cudaB);
*/
  hipFree(answerGPU);
}

int main(){
  srand(time(0));
  vector<string> result;

  for(int i = 0; i < 9; i++){
    cout << "Test " << i << ":" << "N : "  << (1 + pow(10,i)) << endl;
    tests(1 + pow(10,i), result);
  }

  cout << "\nРезультаты всех тестов:\n";
  for (const auto& res : result) {
      cout << res << endl;
  }
  
}
