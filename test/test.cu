#include "hip/hip_runtime.h"
#include <imp.cuh>
#include <iostream>
#include <ctime>

using namespace std;

void tests(int N){
  float *A = new float[N];
  float *B = new float[N];

  for (int i = 0; i < N; ++i) {
      A[i] = (float)(rand()) / (float)(RAND_MAX);
      B[i] = (float)(rand()) / (float)(RAND_MAX);
      //A[i] = static_cast<float>(rand()) / RAND_MAX;
      //B[i] = static_cast<float>(rand()) / RAND_MAX;
  }

  float answerCPU = CPUimplementation(A, B, N); // Вычисления на CPU

  float *cudaA, *cudaB, *answerGPU; // Вычисления на GPU
  hipMalloc(&cudaA, N * sizeof(float));
  hipMalloc(&cudaB, N * sizeof(float));
  hipMalloc(&answerGPU, sizeof(float));

  hipMemcpy(cudaA, A, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(cudaB, B, N * sizeof(float), hipMemcpyHostToDevice);

  const int block_size = 256;
  int number_of_blocks = N/block_size + 1;

  GPUimplementation<<<number_of_blocks, block_size>>>(cudaA, cudaB, answerGPU);
  hipDeviceSynchronize();
  float answerGGPU;
  hipMemcpy(&answerGGPU, answerGPU, sizeof(float), hipMemcpyDeviceToHost);

  // Сравнение результатов
  cout << "CPU: " << answerCPU << " GPU: " << answerGGPU << endl;
  if (abs(answerCPU - answerGGPU) < 1e-5) {
      cout << "Результаты совпадают!" << endl;
  } else {
      cout << "Результаты не совпадают!" << endl;
  }

  // Освобождение ресурсов
  delete[] A;
  delete[] B;
  hipFree(cudaA);
  hipFree(cudaB);
  hipFree(answerGPU);
}

int main(){
  srand(time(0));
  for(int i = 1; i < 100; i++){
    cout << "Test " << i << ":" << endl;
    tests(1000000 + 25000 * i);
  }
}
