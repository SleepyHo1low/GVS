#include "hip/hip_runtime.h"
#include "imp.cuh"

__global__ void GPUimplementation(float* a, float* b, float* result, int N)
{
    __shared__ float partialSums[THREADS_PER_BLOCK];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;

    // Инициализация разделяемой памяти нулями
    partialSums[tid] = 0.0f;

    if (i < N) {
        // Вычисление локальной суммы
        partialSums[tid] = a[i] * b[i];

        __syncthreads();

        // Суммирование результатов в пределах блока
        if (tid == 0) {
            int sum = 0;
            for (int i = 0; i < THREADS_PER_BLOCK; i++) {
                sum += partialSums[i];
            }
            __syncthreads(); // Синхронизация перед атомарной операцией
            atomicAdd(result, sum);
        }
    }
}

float CPUimplementation(float* a, float* b, int N)
{
    float temp = 0.0f;
    for (int i = 0; i < N; ++i)
    {
        temp += a[i] * b[i];
    }

    return temp;
}
