#include "hip/hip_runtime.h"
#include "imp.cuh"

__global__ void GPUimplementation(const float* a, const float* b, float* result, int N) {
    // Каждая нить обрабатывает несколько элементов
    int tid = threadIdx.x;
    int blockDim = blockDim.x;
    int i = blockIdx.x * blockDim + tid;

    // Разделяемая память для блока
    __shared__ float shared_a[blockDim];
    __shared__ float shared_b[blockDim];

    // Копирование данных из глобальной памяти в разделяемую
    shared_a[tid] = a[i];
    shared_b[tid] = b[i];
    __syncthreads();

    // Вычисление локальной суммы
    float sum = 0;
    for (int stride = blockDim/2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sum += shared_a[tid+stride] * shared_b[tid+stride];
        }
        __syncthreads();
    }

    // Запись результата в глобальную память
    if (tid == 0) {
        atomicAdd(result, sum);
    }
}

float CPUimplementation(float* a, float* b, int N)
{
    float temp = 0.0f;
    for (int i = 0; i < N; ++i)
    {
        temp += a[i] * b[i];
    }

    return temp;
}
