#include "hip/hip_runtime.h"
#include "imp.cuh"

__global__ void GPUimplementation(float* a, float* b, float* result)
{
        int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;

    // Âû÷èñëåíèå ëîêàëüíîé ñóììû
    partialSums[tid] = a[i] * b[i];
    __syncthreads();

    // Ñóììèðîâàíèå ðåçóëüòàòîâ â ïðåäåëàõ áëîêà
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            partialSums[tid] += partialSums[tid + s];
        }
        __syncthreads();
    }

    // Çàïèñü ðåçóëüòàòà â ãëîáàëüíóþ ïàìÿòü
    if (tid == 0) {
        atomicAdd(result, partialSums[0]);
    }
}

float CPUimplementation(float* a, float* b, int N)
{
    float temp = 0;
    for (int i = 0; i < N; ++i)
    {
        temp += a[i] * b[i];
    }

    return(temp);
}
