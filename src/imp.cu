#include "hip/hip_runtime.h"
#include "imp.cuh"

__global__ void GPUimplementation(float* a,float* b, float* result, int N) {
    // Каждая нить обрабатывает несколько элементов
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;

    // Разделяемая память для блока
    __shared__ float shared_a[blockDim.x];
    __shared__ float shared_b[blockDim.x];

    // Копирование данных из глобальной памяти в разделяемую
    shared_a[tid] = a[i];
    shared_b[tid] = b[i];
    __syncthreads();

    // Вычисление локальной суммы
    float sum = 0;
    for (int stride = blockDim.x/2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sum += shared_a[tid+stride] * shared_b[tid+stride];
        }
        __syncthreads();
    }

    // Запись результата в глобальную память
    if (tid == 0) {
        atomicAdd(result, sum);
    }
}

float CPUimplementation(float* a, float* b, int N)
{
    float temp = 0.0f;
    for (int i = 0; i < N; ++i)
    {
        temp += a[i] * b[i];
    }

    return temp;
}
