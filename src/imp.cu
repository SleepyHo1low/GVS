#include "hip/hip_runtime.h"
#include "imp.cuh"

__global__ void GPUimplementation(float* a, float* b, float* result, int N)
{
    // Инициализация разделяемой памяти нулями
    partialSums[threadIdx.x] = 0.0f;

    __syncthreads();
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;

    if( i<N){
        // Загрузка данных из глобальной памяти в разделяемую
        float a_local = a[i];
        float b_local = b[i];

        // Вычисление локальной суммы
        partialSums[tid] = a_local * b_local;

        // Суммирование результатов в пределах блока
        __syncthreads();
        for (int s = blockDim.x / 2; s > 0; s >>= 1) {
            if (tid < s) {
                partialSums[tid] += partialSums[tid + s];
            }
            __syncthreads();
        }

        // Запись результата в глобальную память
        if (tid == 0) {
            atomicAdd(result, partialSums[0]);
        }
    }
}

float CPUimplementation(float* a, float* b, int N)
{
    float temp = 0.0f;
    for (int i = 0; i < N; ++i)
    {
        temp += a[i] * b[i];
    }

    return temp;
}
