#include "hip/hip_runtime.h"
#include "imp.cuh"

__global__ void GPUimplementation(float* a, float* b, float* result, int N)
{

    __shared__ float partialSums[TREADS_PER_BLOCK];
    // Инициализация разделяемой памяти нулями
    partialSums[threadIdx.x] = 0.0f;
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;

    if( i<N){

        // Вычисление локальной суммы
        partialSums[tid] = a[i] * b[i];

        __syncthreads();
        // Суммирование результатов в пределах блока
        if( tid ==0){
        
            int sum =0;
            for( int i=0; i<TREADS_PER_BLOCK; i++){
                sum+=partialSums[i];
            }
            atomicAdd(result, sum);
        }
    }
}

float CPUimplementation(float* a, float* b, int N)
{
    float temp = 0.0f;
    for (int i = 0; i < N; ++i)
    {
        temp += a[i] * b[i];
    }

    return temp;
}
