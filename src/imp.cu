#include "hip/hip_runtime.h"
#include "imp.cuh"

__global__ void GPUimplementation(float* a, float* b, float* result)
{
    int i = threadIdx.x + blockDim.x + blockIdx.x;
    atomicAdd(result, a[i] * b[i]);
}

float CPUimplementation(float* a, float* b, int N)
{
    float temp = 0;
    for (int i = 0; i < N; ++i)
    {
        temp += a[i] * b[i];
    }

    return(temp);
}