#include "hip/hip_runtime.h"
#include "imp.cuh"
__shared__ float partialSums[THREADS_PER_BLOCK];

__global__ void GPUimplementation(float* a, float* b, float* result)
{
        int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;

    // ���������� ��������� �����
    partialSums[tid] = a[i] * b[i];
    __syncthreads();

    // ������������ ����������� � �������� �����
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            partialSums[tid] += partialSums[tid + s];
        }
        __syncthreads();
    }

    // ������ ���������� � ���������� ������
    if (tid == 0) {
        atomicAdd(result, partialSums[0]);
    }
}

float CPUimplementation(float* a, float* b, int N)
{
    float temp = 0;
    for (int i = 0; i < N; ++i)
    {
        temp += a[i] * b[i];
    }

    return(temp);
}
