#include "hip/hip_runtime.h"
#include "imp.cuh"

__global__ void GPUatomicimplementation(float* a,float* b, float* result, int N) {
    // Каждый поток вычисляет скалярное произведение для своей части массива
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        atomicAdd(result, a[idx] * b[idx]);
    }
}

__global__ void GPUimplementation(float *a, float *b, float *result, int n) {
    extern __shared__ float sdata[];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;

    if (i < n) {
        sdata[tid] = a[i] * b[i];
    } else {
        sdata[tid] = 0.0f;
    }
    __syncthreads();

    // Редукция в блоке (например, параллельное суммирование)
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(c, sdata[0]);
    }
}

float CPUimplementation(float* a, float* b, int N)
{
    float temp = 0.0f;
    for (int i = 0; i < N; ++i)
    {
        temp += a[i] * b[i];
    }

    return temp;
}
