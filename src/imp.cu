#include "hip/hip_runtime.h"
#include "imp.cuh"

__global__ void GPUimplementation(float* a,float* b, float* result, int N) {
    // Каждый поток вычисляет скалярное произведение для своей части массива
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        atomicAdd(result, a[idx] * b[idx]);
    }
}

float CPUimplementation(float* a, float* b, int N)
{
    float temp = 0.0f;
    for (int i = 0; i < N; ++i)
    {
        temp += a[i] * b[i];
    }

    return temp;
}
