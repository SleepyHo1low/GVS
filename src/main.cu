#include <imp.cuh>
#include <iostream>
#include <ctime>
#include <gtest/gtest.h>

using namespace std;

void tests(int N){
  float *A = new float[N];
  float *B = new float[N];

  for (int i = 0; i < N; ++i) {
      A[i] = static_cast<float>(rand()) / RAND_MAX;
      B[i] = static_cast<float>(rand()) / RAND_MAX;
  }
  
  float answerCPU = CPUimplementation(A, B, N); // Вычисления на CPU

  float *cudaA, *cudaB, *answerGPU; // Вычисления на GPU
  hipMalloc(&cudaA, N * sizeof(float));
  hipMalloc(&cudaB, N * sizeof(float));
  hipMalloc(&answerGPU, sizeof(float));

  hipMemcpy(cudaA, A, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(cudaB, B, N * sizeof(float), hipMemcpyHostToDevice);

  const int block_size = 256;
  int number_of_blocks = (N + block_size - 1) / block_size;

  GPUimplementation<<<number_of_blocks, block_size>>>(cudaA, cudaB, answerGPU);
  hipDeviceSynchronize();
  float answerGGPU;
  hipMemcpy(&answerGGPU, answerGPU, sizeof(float), hipMemcpyDeviceToHost);

  // Сравнение результатов
  ASSERT_NEAR(answerCPU, answerGGPU, 1e-5) << "CPU: " << answerCPU << " GPU: " << answerGGPU;

  // Освобождение ресурсов
  delete[] A;
  delete[] B;
  hipFree(cudaA);
  hipFree(cudaB);
  hipFree(answerGPU);
}

TEST(CpuGpuTests, CompareResults) {
    for (int i = 1; i <= 50; ++i) { // Запускаем 50 тестов с увеличением размера массива
        int N = 10000 + i * 50000; 
        tests(N); // Запуск теста
    }
}

int main(int argc, char **argv){
  srand(time(0));
  ::testing::InitGoogleTest(&argc, argv); 
  return RUN_ALL_TESTS();
}
