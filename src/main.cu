﻿#include <stdio.h>
#include <iostream>
#include <chrono>
#include <ctime>
#include <hiprand.h>
#include "imp.cuh"
#include "data.cpp"

using namespace std;
using namespace std::chrono;

void lineInConsole(char ch) {
    cout << "\n\n";
    for (int i = 0; i < 50; i++) {
        cout << ch;
    }
    cout << "\n";
}
void CPUcalc(float* A, float* B, int N,float &answerCPU) {
    //CPU
    auto start = high_resolution_clock::now();
    answerCPU = CPUimplementation(A, B, N);
    auto stop = high_resolution_clock::now();
    int ms = duration_cast<milliseconds>(stop - start).count();
    int ns = duration_cast<nanoseconds>(stop - start).count();
    cout << "Answer (CPU): " << answerCPU << " time: " <<
        ms << "." << ns - ms * 1000000 << " ms" << endl;
    return;
}

void GPUcalc(float* A, float* B, int N, float& answerGGPU,bool is_atomic) {
    //GPU
    const int floatS = N * sizeof(float);
    float* answerGPU, * cudaA, * cudaB;
    

    hipMalloc(&cudaA, floatS);
    hipMalloc(&cudaB, floatS);
    hipMalloc(&answerGPU, sizeof(float));

    hipMemcpy(cudaA, A, floatS, hipMemcpyHostToDevice);
    hipMemcpy(cudaB, B, floatS, hipMemcpyHostToDevice);

    hipMemset(answerGPU, 0, sizeof(float));

    int number_of_blocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;;

    hipEvent_t startGPU, stopGPU;
    hipEventCreate(&startGPU);
    hipEventCreate(&stopGPU);

    hipEventRecord(startGPU, 0);

    is_atomic ?
        GPUatomicimplementation << < number_of_blocks, THREADS_PER_BLOCK >> > (cudaA, cudaB, answerGPU, N) :
        GPUimplementation << < number_of_blocks, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(float) >> > (cudaA, cudaB, answerGPU, N);
    hipEventRecord(stopGPU, 0);
    hipEventSynchronize(stopGPU);

    // Проверка ошибок
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(error) << std::endl;
        system("pause");
        exit(-1);
    }

    hipMemcpy(&answerGGPU, answerGPU, sizeof(float), hipMemcpyDeviceToHost);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, startGPU, stopGPU);
    is_atomic ?
        cout << "Answer (GPUatomic): " << answerGGPU << " time: " << milliseconds << " ms" << endl :
        cout << "Answer (GPU): " << answerGGPU << " time: " << milliseconds << " ms" << endl;
    hipFree(cudaA);
    hipFree(cudaB);
    hipFree(answerGPU);
    return;
}

void showDevice() {
    int device_count;
    hipGetDeviceCount(&device_count);
    cout << "count GPU device :" << device_count << endl;
    hipDeviceProp_t prop;
    for (int i = 0; i < device_count; i++) {
        cout<< endl << i << ". ";
        hipGetDeviceProperties(&prop, i);
        cout << "Device Name: " << prop.name << std::endl;
        cout << "Compute Capability: " << prop.major << "." << prop.minor
            << endl;
        cout << "Total Global Memory: " << prop.totalGlobalMem / (1024 * 1024) << " MB" << endl;
    }
    return;
}
int main()
{
    lineInConsole('#');
    showDevice();
    lineInConsole('#');
    while (1) {
        cout << "1. DO\n"
            << "2. CLR data\n"
            << "3. EXIT\n";
        string action;
        cout << "Input: ";
        cin >> action;
        if (action == "3") exit(0);
        else if (action == "2") remove("data.bin");
        else if (action == "1") {

            srand(time(0));

            Data data("data.bin");
            const int N = data.n;
            float* A = data.dataA;
            float* B = data.dataB;

            cout << "Num elements: " << N << endl;
            lineInConsole('#');
            float answerCPU;
            CPUcalc(A, B, N, answerCPU);
            lineInConsole('#');
            float answerGGPU;
            GPUcalc(A, B, N, answerGGPU,true);
            cout << "Diff (CPU - GPU): " << answerCPU - answerGGPU << endl;
            lineInConsole('#');
            GPUcalc(A, B, N, answerGGPU, false);
            cout << "Diff (CPU - GPU): " << answerCPU - answerGGPU << endl;
            lineInConsole('#');
        }
        else
            cout << "Incorrect input:" << action;
    }
    return(0);
}
