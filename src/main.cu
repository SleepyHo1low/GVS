#include <stdio.h>
#include <iostream>
#include <chrono>
#include <ctime>
#include <hiprand.h>
#include "imp.cuh"
#include "data.cpp"
using namespace std;
using namespace std::chrono;


int main()
{
    srand(time(0));

    Data data("/content/Data/data.bin");
    const int N = data.n;
    const int floatS = N*sizeof(float);

    float *A = data.dataA;
    float *B = data.dataB;

    float answerCPU;

   

    //CPU
    auto start = high_resolution_clock::now();
    answerCPU = CPUimplementation(A, B, N);
    auto stop = high_resolution_clock::now();

    cout << "Answer (CPU): " << answerCPU << " time: " << duration_cast<milliseconds>(stop - start).count() << " ms" << endl;
    
    //GPU
    float *answerGPU = new float(), *answerGGPU = new float();
    *answerGPU = 0;
    *answerGGPU = 0;
    float* cudaA;
    float* cudaB;

    hipMalloc(&cudaA, floatS);
    hipMalloc(&cudaB, floatS);
    hipMalloc(&answerGPU, sizeof(float));

    hipMemcpy(cudaA, A, floatS, hipMemcpyHostToDevice);
    hipMemcpy(cudaB, B, floatS, hipMemcpyHostToDevice);

    int number_of_blocks = N / THREADS_PER_BLOCK + 1;

    cout<<"THREADS_PER_BLOCK = "<<THREADS_PER_BLOCK<<endl;
    cout<<"number_of_blocks = "<<number_of_blocks<<endl;
    hipEvent_t startGPU, stopGPU;
    hipEventCreate(&startGPU);
    hipEventCreate(&stopGPU);

    hipEventRecord(startGPU);
    GPUimplementation<<< number_of_blocks, THREADS_PER_BLOCK >>>(cudaA, cudaB, answerGPU, N);
    hipDeviceSynchronize();

    
    hipEventRecord(stopGPU);
    // �������� ������
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(error) << std::endl;
        return 1;
    }
    hipMemcpy(answerGGPU, answerGPU, sizeof(float), hipMemcpyDeviceToHost);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, startGPU, stopGPU);

    cout << "Answer (GPU): " << *answerGGPU << " time: " << milliseconds << " ms" << endl;

    hipFree(cudaA);
    hipFree(cudaB);
    hipFree(answerGPU);


    return(0);
}
