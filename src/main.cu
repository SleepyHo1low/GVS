#include <stdio.h>
#include <iostream>
#include <chrono>
#include <ctime>
#include <hiprand.h>
#include "imp.cuh"
#include "data.cpp"
using namespace std;
using namespace std::chrono;


int main()
{
    while (1) {
        cout << "1. DO\n"
            << "2. CLR data\n"
            << "3. EXIT\n";
        string action;
        cout << "Input: ";
        cin >> action;
        if (action == "3") exit(0);
        else if (action == "2") remove("data.bin");
        else if (action == "1") {

            srand(time(0));

            Data data("data.bin");
            const int N = data.n;

            float* A = data.dataA;
            float* B = data.dataB;

            float answerCPU;



            //CPU
            auto start = high_resolution_clock::now();
            answerCPU = CPUimplementation(A, B, N);
            auto stop = high_resolution_clock::now();
            int ms = duration_cast<milliseconds>(stop - start).count();
            int ns = duration_cast<nanoseconds>(stop - start).count();
            cout << "Num elements: " << N << endl;
            cout << "Answer (CPU): " << answerCPU << " time: " <<
                ms << "." << ns - ms * 1000000 << " ms" << endl;
            //cout<<" TEST " << ms <<" " << ns << " TEST" << endl;
            //GPU
            const int floatS = N * sizeof(float);
            float* answerGPU, * cudaA, * cudaB;
            float answerGGPU;

            hipMalloc(&cudaA, floatS);
            hipMalloc(&cudaB, floatS);
            hipMalloc(&answerGPU, sizeof(float));

            hipMemcpy(cudaA, A, floatS, hipMemcpyHostToDevice);
            hipMemcpy(cudaB, B, floatS, hipMemcpyHostToDevice);

            hipMemset(answerGPU, 0, sizeof(float));

            int number_of_blocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;;

            //cout<<"THREADS_PER_BLOCK = "<<THREADS_PER_BLOCK<<endl;
            //cout<<"number_of_blocks = "<<number_of_blocks<<endl;
            hipEvent_t startGPU, stopGPU;
            hipEventCreate(&startGPU);
            hipEventCreate(&stopGPU);

            hipEventRecord(startGPU,0);
            //GPUatomicimplementation << < number_of_blocks, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(float) >> > (cudaA, cudaB, answerGPU, N);
            GPUimplementation << < number_of_blocks, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(float) >> > (cudaA, cudaB, answerGPU, N);
            hipEventRecord(stopGPU,0);
            hipEventSynchronize(stopGPU);

            // �������� ������
            hipError_t error = hipGetLastError();
            if (error != hipSuccess) {
                std::cout << "CUDA error: " << hipGetErrorString(error) << std::endl;
                return 1;
            }

            hipMemcpy(&answerGGPU, answerGPU, sizeof(float), hipMemcpyDeviceToHost);

            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, startGPU, stopGPU);
            
            cout << "Answer (GPU): " << answerGGPU << " time: " << milliseconds << " ms" << endl;
            cout << "Diff (CPU - GPU): " << answerCPU - answerGGPU << endl;
            hipFree(cudaA);
            hipFree(cudaB);
            hipFree(answerGPU);
        }
        else
            cout << "Incorrect input:" << action;
    }
    return(0);
}
