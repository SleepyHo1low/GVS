#include <stdio.h>
#include <iostream>
#include <chrono>
#include <ctime>
#include <hiprand.h>
#include "imp.cuh"
#include "data.cpp"
using namespace std;
using namespace std::chrono;


int main()
{
    srand(time(0));

    Data data("/content/Data/data.bin");
    const int N = data.n;
    const int floatS = N*sizeof(float);

    float *A = data.dataA;
    float *B = data.dataB;

    float answerCPU;

   

    //CPU
    auto start = high_resolution_clock::now();
    answerCPU = CPUimplementation(A, B, N);
    auto stop = high_resolution_clock::now();

    cout << "Answer (CPU): " << answerCPU << " time: " << duration_cast<milliseconds>(stop - start).count() << " ms" << endl;
    
    //GPU
    float *answerGPU = new float(), *answerGGPU = new float();
    *answerGPU = 0;
    *answerGGPU = 0;
    float* cudaA;
    float* cudaB;

    hipMalloc(&cudaA, floatS);
    hipMalloc(&cudaB, floatS);
    hipMalloc(&answerGPU, sizeof(float));

    hipMemcpy(cudaA, A, floatS, hipMemcpyHostToDevice);
    hipMemcpy(cudaB, B, floatS, hipMemcpyHostToDevice);

    const int block_size = 256;
    int number_of_blocks = (N + block_size - 1) / block_size;

    hipEvent_t startGPU, stopGPU;
    hipEventCreate(&startGPU);
    hipEventCreate(&stopGPU);

    hipEventRecord(startGPU);
    GPUimplementation<<< number_of_blocks, block_size >>>(cudaA, cudaB, answerGPU, N);
    hipDeviceSynchronize();

    
    hipEventRecord(stopGPU);
    // �������� ������
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(error) << std::endl;
        return 1;
    }
    hipMemcpy(answerGGPU, answerGPU, sizeof(float), hipMemcpyDeviceToHost);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, startGPU, stopGPU);

    cout << "Answer (GPU): " << *answerGGPU << " time: " << milliseconds << " ms" << endl;

    hipFree(cudaA);
    hipFree(cudaB);
    hipFree(partialSums);
    hipFree(answerGPU);
    hipFree(answerGGPU)



    return(0);
}
