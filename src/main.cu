#include <stdio.h>
#include <iostream>
#include <chrono>
#include <ctime>
#include <hiprand.h>
#include "imp.cuh"
#include "data.cpp"
using namespace std;
using namespace std::chrono;


int main()
{
    srand(time(0));

    Data data("/content/Data/data.bin");
    const int N = data.n;

    float *A = data.dataA;
    float *B = data.dataB;

    float answerCPU;

   

    //CPU
    auto start = high_resolution_clock::now();
    answerCPU = CPUimplementation(A, B, N);
    auto stop = high_resolution_clock::now();

    cout << "Answer (CPU): " << answerCPU << " time: " << duration_cast<milliseconds>(stop - start).count() << " ms" << endl;
    
    //GPU
    const int floatS = N*sizeof(float);
    float *answerGPU, *cudaA, *cudaB;
    float answerGGPU;

    hipMalloc(&cudaA, floatS);
    hipMalloc(&cudaB, floatS);
    hipMalloc(&answerGPU, sizeof(float));

    hipMemcpy(cudaA, A, floatS, hipMemcpyHostToDevice);
    hipMemcpy(cudaB, B, floatS, hipMemcpyHostToDevice);

    hipMemset(answerGPU, 0, sizeof(float));

    int number_of_blocks =  (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;;

    //cout<<"THREADS_PER_BLOCK = "<<THREADS_PER_BLOCK<<endl;
    //cout<<"number_of_blocks = "<<number_of_blocks<<endl;
    hipEvent_t startGPU, stopGPU;
    hipEventCreate(&startGPU);
    hipEventCreate(&stopGPU);

    hipEventRecord(startGPU);
    GPUimplementation<<< number_of_blocks, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(float) >>>(cudaA, cudaB, answerGPU, N);
    //hipDeviceSynchronize();

    hipEventRecord(stopGPU);
    // �������� ������
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(error) << std::endl;
        return 1;
    }

    hipMemcpy(&answerGGPU, answerGPU, sizeof(float), hipMemcpyDeviceToHost);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, startGPU, stopGPU);

    cout << "Answer (GPU): " << answerGGPU << " time: " << milliseconds << " ms" << endl;
    cout << "Diff (CPU - GPU): " << answerCPU - answerGGPU<<endl;
    hipFree(cudaA);
    hipFree(cudaB);
    hipFree(answerGPU);


    return(0);
}
