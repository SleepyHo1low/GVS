#include <stdio.h>
#include <iostream>
#include <chrono>
#include <ctime>
#include <hiprand.h>
#include "imp.cuh"
#include "data.cpp"
using namespace std;
using namespace std::chrono;


int main()
{
    setlocale(LC_ALL,"ru");
    srand(time(0));

    Data data("data.txt");
    const int N = data.n;
    const int floatS = N*sizeof(float);

    float *A = data.dataA;
    float *B = data.dataB;

    float answerCPU, *answerGPU = new float(), *answerGGPU = new float();

    *answerGPU = 0;
    *answerGGPU = 0;

    fillArrays(A, B, N);

    //CPU
    auto start = high_resolution_clock::now();
    answerCPU = CPUimplementation(A, B, N);
    auto stop = high_resolution_clock::now();

    cout << "Answer (CPU): " << answerCPU << " time: " << duration_cast<milliseconds>(stop - start).count() << " ms" << endl;
    
    //GPU

    float* cudaA;
    float* cudaB;

    hipMalloc(&cudaA, floatS);
    hipMalloc(&cudaB, floatS);
    hipMalloc(&answerGPU, sizeof(float));

    hipMemcpy(cudaA, A, floatS, hipMemcpyHostToDevice);
    hipMemcpy(cudaB, B, floatS, hipMemcpyHostToDevice);

    const int block_size = 256;
    int number_of_blocks = N/block_size + 1;

    hipEvent_t startGPU, stopGPU;
    hipEventCreate(&startGPU);
    hipEventCreate(&stopGPU);

    hipEventRecord(startGPU);
    GPUimplementation<<< number_of_blocks, block_size >>>(cudaA, cudaB, answerGPU);
    hipDeviceSynchronize();

    hipEventRecord(stopGPU);

    hipMemcpy(answerGGPU, answerGPU, sizeof(float), hipMemcpyDeviceToHost);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, startGPU, stopGPU);

    cout << "Answer (GPU): " << *answerGGPU << " time: " << milliseconds << " ms" << endl;

    hipFree(cudaA);
    hipFree(cudaB);
    hipFree(answerGPU);

    delete[] A;
    delete[] B;

    return(0);
}
