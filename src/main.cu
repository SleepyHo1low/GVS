#include "hip/hip_runtime.h"
#include <imp.cuh>
#include <iostream>
#include <ctime>

using namespace std;

void tests(int N){
  float *A = new float[N];
  float *B = new float[N];

  for (int i = 0; i < N; ++i) {
      A[i] = static_cast<float>(rand()) / RAND_MAX;
      B[i] = static_cast<float>(rand()) / RAND_MAX;
  }
  
  float answerCPU = CPUimplementation(A, B, N); // Вычисления на CPU

  float *cudaA, *cudaB, *answerGPU; // Вычисления на GPU
  hipMalloc(&cudaA, N * sizeof(float));
  hipMalloc(&cudaB, N * sizeof(float));
  hipMalloc(&answerGPU, sizeof(float));

  hipMemcpy(cudaA, A, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(cudaB, B, N * sizeof(float), hipMemcpyHostToDevice);

  const int block_size = 256;
  int number_of_blocks = (N + block_size - 1) / block_size;

  GPUimplementation<<<number_of_blocks, block_size>>>(cudaA, cudaB, answerGPU);
  hipDeviceSynchronize();
  float answerGGPU;
  hipMemcpy(&answerGGPU, answerGPU, sizeof(float), hipMemcpyDeviceToHost);

  // Сравнение результатов
  cout << "CPU: " << answerCPU << " GPU: " << answerGGPU << endl;
  if (abs(answerCPU - answerGGPU) < 1e-5) {
      cout << "Результаты совпадают!" << endl;
  } else {
      cout << "Результаты не совпадают!" << endl;
  }

  // Освобождение ресурсов
  delete[] A;
  delete[] B;
  hipFree(cudaA);
  hipFree(cudaB);
  hipFree(answerGPU);
}

int main(){
  srand(time(0));
  for(int N = 10000, i = 1; N < 250000000; N += 50000, i++){
    cout << "Test " << i << ":" << endl;
    tests(N);
  }
}
